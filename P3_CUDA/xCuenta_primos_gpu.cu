#include "hip/hip_runtime.h"
/*
Concurrent Computing Practice Code 3: Introduction to CUDA.

Authors:
        - Rivera Martinez Brian 2193036861.
        - Infante Fragoso Jorge 2203025256.

Professor:
        Lopez Jaimes Antonio.

Problem:
        Dado un arreglo de tamaño N con números impares aleatorios, debemos
        regresar cuántos números primos hay.

Restrictions:
        - El tamaño del arreglo debe ser N=10,000,000 o mayor.
        - Los enteros del arreglo deben ser enteros impares mayores a 1,000,000.
*/

/* Bibliotecas a ocupar */
#include "../common/book.h"
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

/* Prototipo de funciones a ocupar */
void llenarArreglo(int *arr, int tam);
void printArreglo(int *arr, int tam);

/* Tamaño del array */
#define N (1 * 100)

/**
 * @brief Valia si un número es primo
 *
 * Esta función valida si todos el número que se le pasa como
 * parámetro es primo.
 *
 * @param n Es el número a validar si es primo.
 * @return 1 en caso de ser primo, 0 en caso contrario.
 */
__device__ int esPrimo(int n) {
  if (n % 3 == 0)
    return 0;

  for (int i = 5; i * i <= n; i = i + 6) {
    if (n % i == 0 || n % (i + 2) == 0) {
      return 0;
    }
  }
  return 1;
}

/**
 * @brief Kernel de marcar si es primo
 *
 * Este kernel marcar en el array siNo con 1 en caso de que el número en el
 * array dev_enteros sea primo, de lo contrario lo marca con un cero.
 *
 * @param dev_eneteros Es el array que se va validar si es primo.
 * @param dev_siNo El array que se va a mapear con unos o ceros para marcar los
 * números primos.
 * @return void.
 */
__global__ void marcarPrimos(int *dev_enteros, int *dev_siNo) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    dev_siNo[tid] = esPrimo(dev_enteros[tid]);
    tid += blockDim.x * gridDim.x;
  }
}

/* Función principal del CPU */
int main(int argc, char const *argv[]) {

  /* Arreglo que tendrá los enteros que queremos verificar si son primos o no */
  int *enteros;
  int *siNo;

  /* Se crean los arreglos en la GPU */
  int *dev_enteros, *dev_siNo;

  /* Inicializar la semilla para generar números aleatorios. */
  srand(time(NULL));

  /* Crear los arreglos de enteros y de resultados (sí/no). */
  enteros = (int *)malloc(N * sizeof(int));
  siNo = (int *)malloc(N * sizeof(int));

  /* Se reserva memoria dinámica en la GPU para los arreglos */
  HANDLE_ERROR(hipMalloc((void **)&dev_enteros, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_siNo, N * sizeof(int)));

  /* Se llena el arreglo 'enteros' con números impares en la CPU */
  llenarArreglo(enteros, N);

  /* Se copia el arreglo 'enteros' y 'siNo' a la GPU */
  HANDLE_ERROR(hipMemcpy(dev_enteros, enteros, N * sizeof(int),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_siNo, siNo, N * sizeof(int), 
                          hipMemcpyHostToDevice));

  /* Se llama a la función de la GPU */
  marcarPrimos<<<32, 1024>>>(dev_enteros, dev_siNo);

  /* Se copia el array 'siNo' de la GPU a la CPU */
  HANDLE_ERROR(hipMemcpy(siNo, dev_siNo, N * sizeof(int), 
                          hipMemcpyDeviceToHost));

  printf("\nArreglo de enteros:\n\n");
  printArreglo(enteros, N);
  printf("\n");
  printf("\nArreglo SiNo\n");
  printArreglo(siNo, N);

  /* Se suman los números que son primos e impares */
  int suma = 0;
  for (int i = 0; i < N; i++)
    suma += siNo[i];

  printf("\nHay %d primos\n\n", suma);

  /* Se libera la memoria dinámica de la CPU */
  free(enteros);
  free(siNo);

  /* Se libera la memoria dinámica de la GPU */
  HANDLE_ERROR(hipFree(dev_enteros));
  HANDLE_ERROR(hipFree(dev_siNo));

  return 0;
}

/**
 * @brief Llena el array
 *
 * Esta función llena el array que se le pasa como parámetro,
 * utilizando números impres.
 *
 * @param arr Es el array que se va a llenar.
 * @param tam Es el tamaño del rray.
 * @return void.
 */
void llenarArreglo(int *arr, int tam) {
  for (int i = 0; i < tam; i++) {
    arr[i] = 10000000 + rand() % 10000;
    arr[i] = 2 * arr[i] + 1;
  }
}

/**
 * @brief Imprime el array.
 *
 * Esta funcíon imprime el arreglo que se le pasa como parámetro.
 *
 * @param arr Es el array que se va a imprimir.
 * @param tam Es el tamaño del array.
 * @return void.
 */
void printArreglo(int *arr, int tam) {
  for (int i = 0; i < tam; i++)
    printf("%d ", arr[i]);
}